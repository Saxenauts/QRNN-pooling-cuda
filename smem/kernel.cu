#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../debug.h"

#define THREADS_PER_BLOCK_X 32
#define THREADS_PER_BLOCK_Y 32

#define MAX_TIME_STEP 30
#define INPUT_DIM 128
#define BATCH_SIZE 1000

/* Macro for index calculations */
#define INDX( time_step, batch_index, col ) ( (time_step + MAX_TIME_STEP * (batch_index + BATCH_SIZE * col)) )

__global__ void fpool_GPU(double* h,  const double* z, const double* f) {

	for(int t = 1; t < MAX_TIME_STEP; t++) {
		// detemine this thread's index in the batch and input dims
		const int mybatch = blockDim.x * blockIdx.x + threadIdx.x;
		const int mycol = blockDim.y * blockIdx.y + threadIdx.y;

		int index = INDX(t, mybatch, mycol);
		int prev_index = INDX(t-1, mybatch, mycol);

		if(mybatch < BATCH_SIZE && mycol < INPUT_DIM) {
			h[index] = f[index]	* h[prev_index] + (1 - f[index]) * z[index];
		}
	}
	return;
}

void fpool_CPU(double* h,  const double* z, const double* f) {
	for(int t = 1; t < MAX_TIME_STEP; t++) {
		for(int row = 0; row < BATCH_SIZE; row++) {
			for(int col = 0; col < INPUT_DIM; col++) {
				int index = INDX(t, row, col);
				int prev_index = INDX(t-1, row, col);
				h[index] = f[index]	* h[prev_index] + (1 - f[index]) * z[index];
			}
		}
	}
	return;
}

int main(int args, char* argv[])
{
	int dev;
	hipDeviceProp_t deviceProp;
	checkCUDA( hipGetDevice( &dev ) );
	checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
	printf("Using GPU %d: %s\n", dev, deviceProp.name );

	// hidden state
	double *h;
	hipMallocManaged(&h, BATCH_SIZE * MAX_TIME_STEP * INPUT_DIM * sizeof(double));
	// convolution outputs
	double *z;
	double *f;
	hipMallocManaged(&z, BATCH_SIZE * MAX_TIME_STEP * INPUT_DIM * sizeof(double));
	hipMallocManaged(&f, BATCH_SIZE * MAX_TIME_STEP * INPUT_DIM * sizeof(double));

	srand(37);
	// initialize conv outputs
	for(int i=0; i<BATCH_SIZE * MAX_TIME_STEP * INPUT_DIM; i++) {
		z[i] = double(rand()) / (double(RAND_MAX) + 1.0);
		f[i] = double(rand()) / (double(RAND_MAX) + 1.0);
	}

	/* Naive GPU Test */

	// set pooling initial state to zero
	memset(h, 0, MAX_TIME_STEP);
	
	dim3 threads( THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1 );
	dim3 blocks( BATCH_SIZE / THREADS_PER_BLOCK_X + 1, 
			   INPUT_DIM / THREADS_PER_BLOCK_Y + 1, 1 );

	float elapsedTime;
	hipEvent_t start, stop;
	checkCUDA( hipEventCreate( &start ) );
	checkCUDA( hipEventCreate( &stop ) );
	checkCUDA( hipEventRecord( start, 0 ) );

	fpool_GPU<<< blocks, threads >>> (h, z, f);
	hipDeviceSynchronize();
	checkKERNEL();

	checkCUDA( hipEventRecord( stop, 0 ) );
	checkCUDA( hipEventSynchronize( stop ) );
	checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );
	fprintf(stdout, "Total time GPU is %f sec\n", elapsedTime / 1000.0f );


	double gpu_sum = 0;
	for(int i=0; i<BATCH_SIZE * MAX_TIME_STEP * INPUT_DIM; i++) {
		gpu_sum += h[i];
	}

	/* CPU Test */
	memset(h, 0, MAX_TIME_STEP);

	checkCUDA( hipEventRecord( start, 0 ) );

	fpool_CPU(h, z, f);

	checkCUDA( hipEventRecord( stop, 0 ) );
	checkCUDA( hipEventSynchronize( stop ) );
	checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );
	fprintf(stdout, "Total time CPU is %f sec\n", elapsedTime / 1000.0f );

	double cpu_sum = 0;
	for(int i=0; i<BATCH_SIZE * MAX_TIME_STEP * INPUT_DIM; i++) {
		cpu_sum += h[i];
	}

	double error = gpu_sum - cpu_sum;
	printf("error is %f\n", error);
	if(error > 10)printf("FAIL\n");
	else printf("PASS\n");

	hipFree(h);
	hipFree(z);
	hipFree(f);

	return 0;
}
